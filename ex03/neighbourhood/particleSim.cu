
#include<iostream>
#include<fstream>
#include<cmath>
#include<string>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<hip/hip_runtime.h>
#include "../param.cpp"
#include "../input.cpp"
#include "vel_verlet.cpp"
#include "../output.cpp"

using namespace std;

void cudasafe(int error, string message, string file, int line) {
    if (error != hipSuccess) {
            cout<<stderr<< " CUDA Error: "<<message<<" : "<<error<<". In "<<file<<" line "<<line<<endl; 
            exit(-1);
    }
}

int main(){
        
    //reading input parameters
    string paramFileName ="blocks_big.par",input_path = "../Question/input/";
    string part_input_file, part_out_name_base, vtk_out_name_base;
    double timeStep, timeEnd, epsilon, sigma;
    unsigned part_out_freq, vtk_out_freq, cl_wg_1dsize;

    unsigned x_n, y_n, z_n, cl_wg_3dsize_x, cl_wg_3dsize_y, cl_wg_3dsize_z;
    double x_min, x_max, y_min, y_max, z_min, z_max,
         r_cut, r_skin;
    
    // reading .par file
    {
        // in param.cpp file
        readParam(
            input_path + paramFileName,
            part_input_file, part_out_name_base, vtk_out_name_base,
            timeStep, timeEnd, epsilon, sigma,
            part_out_freq, vtk_out_freq, cl_wg_1dsize,
            cl_wg_3dsize_x, cl_wg_3dsize_y, cl_wg_3dsize_z,
            x_min, x_max, y_min, y_max, z_min, z_max,
            x_n, y_n, z_n, r_cut, r_skin
        );
        // outParam(
        //     part_input_file, part_out_name_base, 
        //     vtk_out_name_base, timeStep, timeEnd, epsilon, sigma,
        //     part_out_freq, vtk_out_freq, cl_wg_1dsize,
        //     cl_wg_3dsize_x, cl_wg_3dsize_y, cl_wg_3dsize_z,
        //     x_min, x_max, y_min, y_max, z_min, z_max,
        //     x_n, y_n, z_n, r_cut, r_skin
        // );
    }
    
    double del_x=(x_max-x_min)/x_n, del_y=(y_max-y_min)/y_n, del_z=(z_max-z_min)/z_n;
    // declearing host vector memory
    unsigned N, dim, frames = (timeEnd/timeStep);  
    // frames -> # of timeframes
    // N -> # of particles
    // dim -> dimension of vector  
    thrust::host_vector<double> sliced;

    readInput(input_path + part_input_file,sliced,N,dim); // in input.cpp
    host_vector<double> x(N*dim,0), v(N*dim,0), m(N,0),f(N*dim,0) ;

    // extracting m,x,v data from sliced
    {
        extract(
            raw_pointer_cast(&x[0]),
            raw_pointer_cast(&v[0]),
            raw_pointer_cast(&m[0]),
            raw_pointer_cast(sliced.data()), N, dim
        );
        // outInput(
        //     raw_pointer_cast(x.data()),
        //     raw_pointer_cast(v.data()),
        //     raw_pointer_cast(m.data()),
        //     N,dim
        // ); // in input.cpp
    }
    // CUDA Programming
    device_vector<double> d_x(x),d_v(v),d_f(N*dim,0),d_f_old(N*dim,0),zeros(N*dim,0),d_m(m), neighbourhood(N*N,0);
    hipDeviceProp_t deviceProp;
    cudasafe(
        hipGetDeviceProperties(&deviceProp,0),
        "Get device Properties",
        __FILE__, __LINE__
    );
    // cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<endl;

    int blockSize = deviceProp.maxThreadsPerBlock, 
    gridSize = (N/deviceProp.maxThreadsPerBlock)+1;

    // cout<<"Block Size: "<<blockSize<<"\nGrid size: "<<gridSize<<endl
    // <<"Frames: "<<frames<<endl;

    // create neighbourhood list

    calN<<<gridSize, blockSize>>>(
        raw_pointer_cast(&neighbourhood[0]),
        raw_pointer_cast(&d_x[0]),
        r_cut, N, dim
    );

    // Initial force calculation
    calF<<<gridSize, blockSize>>>(
        raw_pointer_cast(&d_x[0]),
        raw_pointer_cast(&d_f[0]),
        raw_pointer_cast(&neighbourhood[0]),
        N, dim, epsilon, sigma, r_cut
    );
    cudasafe(
        hipDeviceSynchronize(),
        "sync threads", 
        __FILE__, __LINE__
    );
    writeOut(
        part_out_name_base, 0,
        raw_pointer_cast(&m[0]),
        raw_pointer_cast(&x[0]),
        raw_pointer_cast(&v[0]),
        N, dim
    ); // in output.cpp
    writeVTK(
        vtk_out_name_base, 0,
        raw_pointer_cast(&m[0]),
        raw_pointer_cast(&x[0]),
        raw_pointer_cast(&v[0]),
        N, dim
    ); // in output.cpp

    for(int i=1; i<=frames; i++){

        calX<<<gridSize, blockSize>>>(
            raw_pointer_cast(&d_x[0]),
            raw_pointer_cast(&d_v[0]),
            raw_pointer_cast(&d_f[0]),
            raw_pointer_cast(&d_m[0]), 
            timeStep, N, dim,
            x_min, x_max, y_min, 
            y_max, z_min, z_max
        );
        cudasafe(
            hipDeviceSynchronize(),
            "sync threads", 
            __FILE__, __LINE__
        );

        d_f_old = d_f;
        thrust::copy(zeros.begin(), zeros.end(), d_f.begin());
        
        calN<<<gridSize, blockSize>>>(
            raw_pointer_cast(&neighbourhood[0]),
            raw_pointer_cast(&d_x[0]),
            r_cut, N, dim
        );

        calF<<<gridSize, blockSize>>>(
            raw_pointer_cast(&d_x[0]),
            raw_pointer_cast(&d_f[0]),
            raw_pointer_cast(&neighbourhood[0]),
            N, dim, epsilon, sigma, r_cut
        );

        cudasafe(
            hipDeviceSynchronize(),
            "sync threads", 
            __FILE__, __LINE__
        );

        calV<<<gridSize,blockSize>>>(
            raw_pointer_cast(&d_v[0]),
            raw_pointer_cast(&d_f[0]),
            raw_pointer_cast(&d_f_old[0]),
            raw_pointer_cast(&d_m[0]),
            timeStep, N, dim
        );

        cudasafe(
            hipDeviceSynchronize(),
            "sync threads", 
            __FILE__, __LINE__
        );
        
        if(i%part_out_freq == 0){
            m = d_m; x = d_x; v = d_v;
            writeOut(
                part_out_name_base, (i/part_out_freq),
                raw_pointer_cast(&m[0]),
                raw_pointer_cast(&x[0]),
                raw_pointer_cast(&v[0]),
                N, dim
            ); // in output.cpp
        }
        if(i%vtk_out_freq == 0){
            m = d_m; x = d_x; v = d_v;
            writeVTK(
                part_out_name_base, (i/vtk_out_freq),
                raw_pointer_cast(&m[0]),
                raw_pointer_cast(&x[0]),
                raw_pointer_cast(&v[0]),
                N, dim
            ); // in output.cpp
        }
    }

    std::cout<<"\n\nAll done!\n\n";
        
    return 0;
}
#include<iostream>
#include <hip/hip_runtime.h>
#include<cmath>

using namespace std;

int w = 2048, h = w;
int d = 64;

void cudasafe(int error, string message="(---)", string file = "(-this file-)", int line = -1) {
    if (error != hipSuccess) {
            cout<<stderr<< " CUDA Error: "<<message<<" : "<<error<<". In "<<file<<" line "<<line<<endl; 
            exit(-1);
    }
}
__global__ void set(int* val,const int w,const int d){
    int idx = blockIdx.x * (w/d)*(w/d) + threadIdx.y * (w/d) + threadIdx.x;
    val[idx] = idx;
}

int main(){
    
    int* d_int;
    int* val = new int [w*h];
    cudasafe(hipMalloc(&d_int,w*h*sizeof(int)),"Mem Allo",__FILE__,__LINE__);
    dim3 blocks(w/d,h/d);
    dim3 grids(d*d);
    set<<<grids, blocks>>>(d_int, w,d);
    cudasafe(hipMemcpy(val,d_int,w*h*sizeof(int),hipMemcpyDeviceToHost),
                "Mem Allo",__FILE__,__LINE__);
    for(int i=0; i<w*h; i++)cout<<val[i]<<"  ";
    hipFree(d_int);
    delete[] val;
    return 0;
}